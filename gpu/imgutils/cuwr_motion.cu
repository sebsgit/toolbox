#include "hip/hip_runtime.h"
#include "cuwr_imgdata_priv.h"

typedef unsigned char uchar;

class CoordCalc{
public:
    __device__ CoordCalc(const dim3& thIdx,
                         const dim3& blockId,
                         const dim3& blockDim,
                         const dim3& gridDim)
        :threadIdx_(thIdx)
        ,blockIdx_(blockId)
        ,blockDim_(blockDim)
        ,gridDim_(gridDim)
    {
        const size_t idInBlock = threadIdx_.x + threadIdx_.y*blockDim_.x;
        const size_t blockIdInGrid = blockIdx_.x + blockIdx_.y*gridDim_.x;
        this->pixelId_ = idInBlock + blockIdInGrid*blockDim_.x*blockDim_.y;
    }
    __device__ size_t pixelId() const{
        return this->pixelId_;
    }
    __device__ void getCoords(const size_t width, size_t * row, size_t * col) const{
        *col = (pixelId_) % width;
        *row = (pixelId_) / width;
    }
    __device__ size_t dataOffset(const cuwr_image_kernel_data_t * header) const{
        const size_t col_id = (pixelId_) % header->width;
        const size_t row_id = (pixelId_) / header->width;
        return col_id*header->bpp + row_id*header->widthStep;
    }
    __device__ size_t dataOffset(const size_t row, const size_t col, const cuwr_image_kernel_data_t * header) const{
        return col*header->bpp + row*header->widthStep;
    }
private:
    const dim3& threadIdx_;
    const dim3& blockIdx_;
    const dim3& blockDim_;
    const dim3& gridDim_;
    size_t pixelId_;
};

__device__ float MAD_helper(const dim3& threadIdx,
                            const dim3& blockIdx,
                            const dim3& blockDim,
                            const dim3& gridDim,
                            const uchar * image1,
                            const cuwr_image_kernel_data_t * header1,
                            const uchar * image2,
                            const cuwr_image_kernel_data_t * header2,
                            const cuwr_dim2 * offsets)
            {
                float result = 255.0f*header1->bpp;
                const CoordCalc calc(threadIdx,blockIdx,blockDim,gridDim);
                int row = threadIdx.y + blockIdx.y*blockDim.y;
                int col= threadIdx.x + blockIdx.x*blockDim.x;
                if (row < header1->height && col < header1->width){
                    const uchar * pxIn1 = image1 + calc.dataOffset(row,col,header1);
                    row -= offsets->y;
                    col -= offsets->x;
                    if (row >= 0 && col >=0 && row < header2->height && col < header2->width){
                        const uchar * pxIn2 = image2 + calc.dataOffset(row,col,header2);
                        result = abs((float)pxIn1[0]-(float)pxIn2[0]);
                        if (header1->bpp > 1){
                            result += abs((float)pxIn1[1]-(float)pxIn2[1]);
                            result += abs((float)pxIn1[2]-(float)pxIn2[2]);
                        }
                    }
                }
                return result;
            }

extern "C"{

/* calculate MAD of two images (mean absolute difference) per block
   output: array of MAD values (one number per block)

   this function should be launched with assigned shared memory for the block MAD value
   thread block size should match the logical image block size
*/
__global__ void cuwr_MAD(const uchar * image1, const cuwr_image_kernel_data_t * header1,
                         const uchar * image2, const cuwr_image_kernel_data_t * header2,
                         const cuwr_dim2 * offsets,
                         cuwr_mad_result_t * output)
            {
                __shared__ float block_mad_value;
                const float thisThreadMad = MAD_helper(threadIdx,blockIdx,blockDim,gridDim,
                                                 image1,header1,
                                                 image2,header2,
                                                 offsets);
                if (thisThreadMad > 0.0f)
                    atomicAdd(&block_mad_value,thisThreadMad);
                __syncthreads();
                if (threadIdx.x==0 && threadIdx.y==0){
                    block_mad_value /= blockDim.x*blockDim.y;
                    cuwr_mad_result_t * out = output + blockIdx.x + blockIdx.y*gridDim.x;
                    if (out->madValue > block_mad_value){
                        out->madValue = block_mad_value;
                        out->offset = *offsets;
                    }
                }
            }

/* implementation of Three-Step Search algorithm
*/
__global__ void cuwr_three_step_search(const uchar * image1, const cuwr_image_kernel_data_t * header1,
                                       const uchar * image2, const cuwr_image_kernel_data_t * header2,
                                       const size_t S,
                                       cuwr_dim2 * off,
                                       cuwr_mad_result_t * output)
            {
                extern __shared__ float block_mad[];
                const int blockId = blockIdx.x + blockIdx.y*gridDim.x;
                const int diff_x = off[blockId].x;
                const int diff_y = off[blockId].y;
                const cuwr_dim2 offsets[9] = { cuwr_dim2(-S+diff_x,-S+diff_y),
                                               cuwr_dim2(0+diff_x,-S+diff_y),
                                               cuwr_dim2(S+diff_x,-S+diff_y),
                                               cuwr_dim2(-S+diff_x,0+diff_y),
                                               cuwr_dim2(0+diff_x,0+diff_y),
                                               cuwr_dim2(S+diff_x,0+diff_y),
                                               cuwr_dim2(-S+diff_x,S+diff_y),
                                               cuwr_dim2(0+diff_x,S+diff_y),
                                               cuwr_dim2(S+diff_x,S+diff_y)};
                for (int i=0 ; i<9 ; ++i){
                    const float mad = MAD_helper(threadIdx,blockIdx,blockDim,gridDim,
                                                 image1,header1,
                                                 image2,header2,
                                                 &offsets[i]);
                    if (mad > 0.0f)
                        atomicAdd(&block_mad[i],mad);
                }
                __syncthreads();
                if (threadIdx.x==0 && threadIdx.y==0){
                    for (int i=0 ; i<9 ; ++i){
                        block_mad[i] /= blockDim.x*blockDim.y;
                        cuwr_mad_result_t * out = output + blockId;
                        if (out->madValue > block_mad[i]){
                            out->madValue = block_mad[i];
                            out->offset.x = offsets[i].x;
                            out->offset.y = offsets[i].y;
                            off[blockId].x = offsets[i].x;
                            off[blockId].y = offsets[i].y;
                        }
                    }
                }
            }

/* calculate absolute value of difference between [image1] and [image2+offset]
    that is outp(x,y) = abs(image1(x,y) - image2(x+off.x, y+off.y))
    equal image sizes are assumed
    coordinates out of bounds are assumed to have value = 0
*/
__global__ void cuwr_diff_image( const uchar * image1, const cuwr_image_kernel_data_t * header1,
                                 const uchar * image2, const cuwr_image_kernel_data_t * header2,
                                 const cuwr_dim2 * offsets,
                                 uchar * output, const cuwr_image_kernel_data_t * header_out)
            {
                const CoordCalc calc(threadIdx,blockIdx,blockDim,gridDim);
                const size_t id = calc.pixelId();
                if (id < header1->width*header1->height){
                    size_t row, col;
                    calc.getCoords(header1->width,&row,&col);
                    const uchar * pxIn1 = image1 + calc.dataOffset(header1);
                    uchar * pxOut = output + calc.dataOffset(header_out);
                    row -= offsets->y;
                    col -= offsets->x;
                    if (row < header2->height && col < header2->width){
                        const uchar * pxIn2 = image2 + calc.dataOffset(row,col,header2);
                        pxOut[0] = abs(pxIn1[0]-pxIn2[0]);
                        if (header1->bpp > 1){
                            pxOut[1] = abs(pxIn1[1]-pxIn2[1]);
                            pxOut[2] = abs(pxIn1[2]-pxIn2[2]);
                        }
                    } else{
                        pxOut[0] = pxIn1[0];
                        if (header1->bpp > 1){
                            pxOut[1] = pxIn1[1];
                            pxOut[2] = pxIn1[2];
                        }
                    }
                }
            }

}
