#include "hip/hip_runtime.h"
#include "cuwr_imgdata_priv.h"

/*
  image processing gpu utilities
  this module is intended to internal use in cuwr::Image class
*/

extern "C"
{

/* linear mapping of thread position in grid to a pixel number */
__device__ size_t px_id(const dim3 threadIdx,
                        const dim3 blockIdx,
                        const dim3 blockDim,
                        const dim3 gridDim)
            {
                const size_t idInBlock = threadIdx.x + threadIdx.y*blockDim.x;
                const size_t blockIdInGrid = blockIdx.x + blockIdx.y*gridDim.x;
                return idInBlock + blockIdInGrid*blockDim.x*blockDim.y;
            }
/* find data pointer offset for pixel number pxId */
__device__ int px_offset(const size_t pxId,
                         const cuwr_image_kernel_data_t * header)
            {
                const int col_id = pxId % header->width;
                const int row_id = pxId / header->width;
                return col_id*header->bpp + row_id*header->widthStep;
            }

/* sets pixel value to specified [r,g,b] tripple.
    in case of grayscale images, only 'r' value is used
 */
__global__ void cuwr_set_pixels(unsigned char * data,
                                const cuwr_image_kernel_data_t * header,
                                const size_t offset,
                                const unsigned char r,
                                const unsigned char g,
                                const unsigned char b)
            {
                const size_t pxId = px_id(threadIdx,blockIdx,blockDim,gridDim) + offset;
                if (pxId < header->width*header->height){
                    unsigned char * pxDataPtr = data + px_offset(pxId,header);
                    pxDataPtr[0] = r;
                    if (header->bpp > 1){
                        pxDataPtr[1] = g;
                        pxDataPtr[2] = b;
                    }
                }
            }

/* swaps r<->b channels */
__global__ void cuwr_swap_rgb(unsigned char * data,
                              const cuwr_image_kernel_data_t * header,
                              const size_t offset
                            )
			{
                const size_t pxId = px_id(threadIdx,blockIdx,blockDim,gridDim) + offset;
                if (pxId < header->width*header->height){
                    unsigned char * pxDataPtr = data + px_offset(pxId,header);
                    const unsigned char r = pxDataPtr[0];
                    pxDataPtr[0] = pxDataPtr[2];
                    pxDataPtr[2] = r;
                }
            }
	
}
